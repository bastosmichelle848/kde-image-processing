#include "hip/hip_runtime.h"
// kde_cuda.cu - Implementacao com CUDA em C++

#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

const int WIDTH = 512, HEIGHT = 512;
const float BANDWIDTH = 1.0f;

__device__ float gaussian_kernel(float x, float y, float bandwidth) {
    return expf(-(x*x + y*y) / (2 * bandwidth * bandwidth)) / (2 * M_PI * bandwidth * bandwidth);
}

__global__ void kde_kernel(unsigned char *input, float *output, int width, int height, float bandwidth) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < height && j < width) {
        float density = 0.0f;
        for (int k = 0; k < height; k++) {
            for (int l = 0; l < width; l++) {
                float x_diff = j - l;
                float y_diff = i - k;
                density += input[k * width + l] * gaussian_kernel(x_diff, y_diff, bandwidth);
            }
        }
        output[i * width + j] = density;
    }
}

int main() {
    Mat img = imread("../image.png", IMREAD_GRAYSCALE);
    resize(img, img, Size(WIDTH, HEIGHT));
    
    unsigned char *d_input;
    float *d_output;
    
    hipMalloc(&d_input, WIDTH * HEIGHT);
    hipMalloc(&d_output, WIDTH * HEIGHT * sizeof(float));
    hipMemcpy(d_input, img.data, WIDTH * HEIGHT, hipMemcpyHostToDevice);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x, (HEIGHT + blockSize.y - 1) / blockSize.y);
    kde_kernel<<<gridSize, blockSize>>>(d_input, d_output, WIDTH, HEIGHT, BANDWIDTH);
    
    float output[WIDTH * HEIGHT];
    hipMemcpy(output, d_output, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
    
    FileStorage file("output_cuda.yml", FileStorage::WRITE);
    file << "output" << Mat(HEIGHT, WIDTH, CV_32F, output);
    file.release();
    
    hipFree(d_input);
    hipFree(d_output);
    
    cout << "Resultados salvos em 'output_cuda.yml'." << endl;
    return 0;
}
